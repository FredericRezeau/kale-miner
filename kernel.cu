#include "hip/hip_runtime.h"
/*
    MIT License
    Author: Fred Kyung-jin Rezeau <fred@litemint.com>, 2024
    Permission is granted to use, copy, modify, and distribute this software for any purpose
    with or without fee.
    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <cstring>
#include <cstddef>

#include "utils/keccak.cuh"

constexpr int maxDataSize = 256;
__constant__ std::uint8_t deviceData[maxDataSize];

#define CUDA_CALL(call)                                                \
    do {                                                               \
        hipError_t err = call;                                        \
        if (err != hipSuccess) {                                      \
            fprintf(stderr, "CUDA Error in %s, line %d: %s\n",         \
                    __FILE__, __LINE__, hipGetErrorString(err));      \
            exit(EXIT_FAILURE);                                        \
        }                                                              \
    } while (0)

__device__ __forceinline__ void updateNonce(std::uint64_t val, std::uint8_t* buffer) {
    #pragma unroll 8
    for (int i = 0; i < 8; i++) {
        buffer[7 - i] = static_cast<std::uint8_t>(val >> (i * 8) & 0xFF);
    }
}

__device__ __forceinline__ bool check(const std::uint8_t* hash, int difficulty) {
    int zeros = 0;
    #pragma unroll 32
    for (int i = 0; i < 32; ++i) {
        int zero = -(hash[i] == 0);
        zeros += (zero & 2) | (~zero & ((-((hash[i] >> 4) == 0)) & 1));
        i += ((hash[i] != 0) | (zeros >= difficulty)) * (32 - i);
    }
    return zeros == difficulty;
}

__device__ __forceinline__ void vCopy(std::uint8_t* dest, const std::uint8_t* src, int size) {
    // Align then copy 8 bytes at a time, more efficient than memcpy.
    int i = 0;
    while (i < size && ((uintptr_t)(dest + i) % 8 != 0)) {
        dest[i] = src[i];
        i++;
    }
    #pragma unroll
    for (; i + 7 < size; i += 8) {
        *(reinterpret_cast<std::uint64_t*>(dest + i)) = *(reinterpret_cast<const std::uint64_t*>(src + i));
    }
    #pragma unroll
    for (; i < size; ++i) {
        dest[i] = src[i];
    }
}

__global__ void run(int dataSize, std::uint64_t startNonce, int nonceOffset, std::uint64_t batchSize, int difficulty,
                                 int* __restrict__ found, std::uint8_t* __restrict__ output, std::uint64_t* __restrict__ validNonce) {
    std::uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    std::uint64_t stride = gridDim.x * blockDim.x;
    if (dataSize > maxDataSize || idx >= batchSize || atomicAdd(found, 0) == 1)
        return;
    std::uint64_t nonceEnd = startNonce + batchSize;
    std::uint8_t threadData[maxDataSize];
    vCopy(threadData, deviceData, dataSize);

    // Nonce distribution is based on thread id - spaced by stride.
    for (std::uint64_t nonce = startNonce + idx; nonce < nonceEnd; nonce += stride) {
        updateNonce(nonce, &threadData[nonceOffset]);
        std::uint8_t hash[32];
        keccak256(threadData, dataSize, hash);
        if (check(hash, difficulty)) {
            if (atomicCAS(found, 0, 1) == 0) {
                memcpy(output, hash, 32);
                atomicExch(reinterpret_cast<unsigned long long int*>(validNonce), static_cast<unsigned long long int>(nonce));
            }
            return;
        }
        if (atomicAdd(found, 0) == 1)
            return;
    }
}

extern "C" int executeKernel(int deviceId, std::uint8_t* data, int dataSize, std::uint64_t startNonce, int nonceOffset, std::uint64_t batchSize,
    int difficulty, int threadsPerBlock, std::uint8_t* output, std::uint64_t* validNonce, bool showDeviceInfo) {
    std::uint8_t* deviceOutput;
    std::size_t outputSize = 32 * sizeof(std::uint8_t);
    int found = 0;
    int* deviceFound;
    std::uint64_t* deviceNonce;
    hipDeviceProp_t deviceProp;
    CUDA_CALL(hipSetDevice(deviceId));
    CUDA_CALL(hipGetDeviceProperties(&deviceProp, deviceId));
    CUDA_CALL(hipMalloc((void**)&deviceFound, sizeof(int)));
    CUDA_CALL(hipMemcpy(deviceFound, &found, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceData), data, dataSize));
    CUDA_CALL(hipMalloc((void**)&deviceOutput, outputSize));
    CUDA_CALL(hipMalloc((void**)&deviceNonce, sizeof(std::uint64_t)));
    CUDA_CALL(hipMemset(deviceNonce, 0, sizeof(std::uint64_t)));

    if (showDeviceInfo) {
        printf("Device: %s\n", deviceProp.name);
        printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("Max threads/blocks: %d\n", deviceProp.maxThreadsPerBlock);
        printf("Max grid size: [%d, %d, %d]\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    }

    int threads = threadsPerBlock;
    std::uint64_t blocks = (batchSize + threads - 1) / threads;
    if (blocks > deviceProp.maxGridSize[0]) {
        blocks = deviceProp.maxGridSize[0];
    }
    std::uint64_t adjustedBatchSize = blocks * threads;
    run<<<(unsigned int)blocks, threads>>>(dataSize, startNonce,
        nonceOffset, adjustedBatchSize, difficulty, deviceFound, deviceOutput, deviceNonce);
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipMemcpy(output, deviceOutput, outputSize, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(&found, deviceFound, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(validNonce, deviceNonce, sizeof(std::uint64_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(deviceOutput));
    CUDA_CALL(hipFree(deviceFound));
    CUDA_CALL(hipFree(deviceNonce));
    return found;
}
